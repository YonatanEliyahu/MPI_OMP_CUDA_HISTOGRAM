#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

#define NUM_OF_THREADS 20
#define NUM_OF_BLOCKS 10

__global__ void calcHistogram(const int *arr, int *histogram, int numElements, int numElementsPerThread)
{
    int tid = (blockDim.x * blockIdx.x + threadIdx.x); // computing thread id
    int from = tid * numElementsPerThread;             // computing start of the scope of work for the current thread
    int to = from + numElementsPerThread;              // computing end of the scope of work for the current thread
    if (to > numElements)                              // if the scope calculation is bigger then the real size of the arr
        to = numElements;                              // end scope in the end of the arr

    for (int index = from; index < to; index++)
    {
        atomicAdd(&histogram[tid * NUMS_IN_RANGE + arr[index]], 1); // increasing the sub-histogram of the current tid
    }
}

__global__ void sumHistogram(const int *histogram, int *collective, int numOfSubArrs)
{
    // the folowing function will get huge histogram arr, and will set a collective histogram arr by suming every index of the sub-histogram
    int tid = threadIdx.x;
    for (int index = 0; index < numOfSubArrs; index++)
        collective[tid] += histogram[tid + index * NUMS_IN_RANGE];
}

int *computeHistogramOnGPU(int *data, int numElements)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(int);

    // Allocate memory on GPU to copy the data from the host
    int *d_A;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //  Copy the data from host to the GPU memory.
    err = hipMemcpy(d_A, data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data array from host to device -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    size_t histogramSize = NUMS_IN_RANGE * NUM_OF_BLOCKS * NUM_OF_THREADS * sizeof(int);
    // Allocate memory on GPU for the histogram results and setting it to 0
    int *h_A;
    err = hipMalloc((void **)&h_A, histogramSize);
    err = hipMemset(h_A, 0, histogramSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Kernel
    int threadsPerBlock = NUM_OF_THREADS;
    int blocksPerGrid = NUM_OF_BLOCKS;
    int numElementsPerThread = numElements / (threadsPerBlock * blocksPerGrid); // calculating the scope of work for each thread
    if (numElements % (threadsPerBlock * blocksPerGrid) != 0)
        numElementsPerThread++; // dealling with leftovers

    calcHistogram<<<blocksPerGrid, threadsPerBlock>>>(d_A, h_A, numElements, numElementsPerThread);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory on GPU for the histogram collective result and setting it to 0
    int realHistogramSize = NUMS_IN_RANGE * sizeof(int);
    int *hcr_A = NULL; // histogram collective result
    err = hipMalloc((void **)&hcr_A, realHistogramSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemset(hcr_A, 0, realHistogramSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Kernel
    threadsPerBlock = NUMS_IN_RANGE;
    blocksPerGrid = 1;

    sumHistogram<<<blocksPerGrid, threadsPerBlock>>>(h_A, hcr_A, NUM_OF_THREADS * NUM_OF_BLOCKS);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the data from GPU to the host memory.
    int *collective_histogram = (int *)malloc(realHistogramSize);
    err = hipMemcpy(collective_histogram, hcr_A, realHistogramSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_A) != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(h_A) != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free allocated memory on GPU
    if (hipFree(hcr_A) != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return collective_histogram;
}
